#include "hip/hip_runtime.h"
#if defined(__HIPCC__) || !defined(__device__)
#include <hip/hip_math_constants.h>
#include "hip/hip_runtime.h"
#include "network.h"

using namespace network_package;

#ifdef __INTELLISENSE__
#define __CUDA_ARCH__ 800
#endif
#define CUDA_CALL(call)                                                         \
	{                                                                           \
		hipError_t err = call;                                                 \
		if (err != hipSuccess) {                                               \
			fprintf(stderr, "CUDA error in file '%s' in line %i: %s.\n",        \
					__FILE__, __LINE__, hipGetErrorString(err));               \
			exit(EXIT_FAILURE);                                                 \
		}                                                                       \
	}

__constant__ double CONSTANT_PI = 3.1415926535897931;

static double* d_phee_minus_alpha_list_gfx = nullptr;
static double* d_gain_RX_grid_gfx = nullptr;
static double* d_pathloss_list_gfx = nullptr;

static double* host_hmatrix_gfx = nullptr;
static double* d_hmatrix_gfx = nullptr;

static Polar_Coordinates* d_polar_data_gfx = nullptr;
static Polar_Coordinates* host_polar_gfx = nullptr;

static size_t d_malloc_size_gfx = 0;

static double* d_phee_minus_alpha_list_sim = nullptr;
static double* d_gain_RX_grid_sim = nullptr;
static double* d_pathloss_list_sim = nullptr;

static double* host_hmatrix_sim = nullptr;
static double* d_hmatrix_sim = nullptr;

static Polar_Coordinates* d_polar_data_sim = nullptr;
static Polar_Coordinates* host_polar_sim = nullptr;

static size_t d_malloc_size_sim = 0;

__host__ void CUDA_GINIT(const size_t& malloc_size)
{
	hipMalloc(&d_phee_minus_alpha_list_gfx, malloc_size * sizeof(double));
	hipMalloc(&d_gain_RX_grid_gfx, malloc_size * sizeof(double));
	hipMalloc(&d_pathloss_list_gfx, malloc_size * sizeof(double));
	hipHostMalloc(&host_hmatrix_gfx, malloc_size * sizeof(double));
	hipMalloc(&d_hmatrix_gfx, malloc_size * sizeof(double));
	hipMalloc(&d_polar_data_gfx, malloc_size * sizeof(Polar_Coordinates));
	d_malloc_size_gfx = malloc_size;
}

__host__ void CUDA_INIT(const size_t& malloc_size)
{
	hipMalloc(&d_phee_minus_alpha_list_sim, malloc_size * sizeof(double));
	hipMalloc(&d_gain_RX_grid_sim, malloc_size * sizeof(double));
	hipMalloc(&d_pathloss_list_sim, malloc_size * sizeof(double));
	hipHostMalloc(&host_hmatrix_sim, malloc_size * sizeof(double));
	hipMalloc(&d_hmatrix_sim, malloc_size * sizeof(Polar_Coordinates));
	hipMalloc(&d_polar_data_sim, malloc_size * sizeof(Polar_Coordinates));
	d_malloc_size_sim = malloc_size;
}

__host__ void CUDA_GFREE()
{
	hipFree(d_phee_minus_alpha_list_gfx);
	hipFree(d_gain_RX_grid_gfx);
	hipFree(d_pathloss_list_gfx);
	hipHostFree(host_hmatrix_gfx);
	hipFree(d_hmatrix_gfx);
	hipFree(d_polar_data_gfx);
}

__host__ void CUDA_FREE()
{
	hipFree(d_phee_minus_alpha_list_sim);
	hipFree(d_gain_RX_grid_sim);
	hipFree(d_pathloss_list_sim);
	hipHostFree(host_hmatrix_sim);
	hipFree(d_hmatrix_sim);
	hipFree(d_polar_data_sim);
}

/* update the antenna array from updated power and scan angle */
__global__ void antenna_update_kernel(
	const size_t size,
	const double& scan_angle,
	const unsigned& panel_count,
	double* d_phee_minus_alpha_list,
	double* d_gain_RX_grid,
	double* d_pathloss_list,
	double* d_hmatrix)
{
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

	/* update the antenna gain Gtx */
	if (idx < size)
	{
		double phee = (d_phee_minus_alpha_list[idx] + scan_angle) / 2;

		double sin_term = panel_count * sinf(phee);
		double gain_factor_antenna_system = d_gain_RX_grid[idx]; // xN antennas already

		if (sin_term != 0)
		{
			double pow_base = sinf(panel_count * phee) / sin_term;
			gain_factor_antenna_system *= pow_base * pow_base;
		}

		/* update the channel matrix */
		d_hmatrix[idx] = gain_factor_antenna_system / d_pathloss_list[idx];
	}
}

/*update the antenna array from updated powerand scan angle */
__host__ void AAntenna::update(
	const size_t& malloc_size,
	double* phee_minus_alpha_list,
	double* gain_RX_grid,
	double* pathloss_list,
	double* gpu_hmatrix,
	double* host_hmatrix)
{
	int threadsPerBlock = 256;
	int blocksPerGrid = (malloc_size + threadsPerBlock - 1) / threadsPerBlock;

	hipStream_t stream;
	hipStreamCreate(&stream);

	antenna_update_kernel <<< blocksPerGrid, threadsPerBlock >>> (
		malloc_size, current.alpha, current.panel_count, phee_minus_alpha_list, gain_RX_grid, pathloss_list, gpu_hmatrix);

	hipMemcpyAsync(host_hmatrix, gpu_hmatrix, malloc_size * sizeof(double), hipMemcpyDeviceToHost, stream);
	hipStreamSynchronize(stream);
	hipStreamDestroy(stream);

	// Check for any errors launching the kernel
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "antenna_update_kernel() launch failed: %s\n", hipGetErrorString(cudaStatus));
		exit(-1);
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		exit(-1);
	}
}



/* re-calc the signal outs to handsets only (before calling update!) */
__global__ void antenna_init_kernel(
	const size_t data_size,
	const double& current_lambda,
	const double& current_spacing,
	const double& theta_c,
	const unsigned& panel_count,
	const antennadim& antenna_dims,
	double* d_phee_minus_alpha_list,
	double* d_pathloss_list,
	double* d_gain_RX_grid,
	Polar_Coordinates* d_polar_data)
{
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < data_size)
	{
		const double& pioverlambda = CONSTANT_PI / current_lambda;
		const double& phee_temp = 2 * current_spacing * pioverlambda;
		const double& pl_temp_meters = 4 * pioverlambda;
		const double& antenna_dim_factor = 10 * antenna_dims.x * antenna_dims.y / (current_lambda * current_lambda);
		double m_factor = antenna_dims.x * pioverlambda;

		auto& cell_polar_data = d_polar_data[idx];
		double theta_minus_thetaC = cell_polar_data.theta - theta_c;
		double m = m_factor * sinf(theta_minus_thetaC);
		double pow_base = (1 + cached::cos(theta_minus_thetaC)) / 2;
		double singleant_gain = antenna_dim_factor * pow_base * pow_base; // pow(pow_base, 2) equivalent on HOST

		if (m != 0)
		{
			singleant_gain *= pow(cached::sin(m) / m, 2);
		}

		d_phee_minus_alpha_list[idx] = phee_temp * cached::sin(theta_minus_thetaC);
		d_pathloss_list[idx] = pow(pl_temp_meters * cell_polar_data.hype, 2);
		d_gain_RX_grid[idx] = singleant_gain * panel_count;
	}
}

__host__ void AAntenna::init(
	const size_t& d_malloc_size,
	double* d_phee_minus_alpha_list,
	double* d_pathloss_list,
	double* d_gain_RX_grid,
	const Polar_Coordinates* d_polar_data)
{
	int threadsPerBlock = 256;
	int blocksPerGrid = (d_malloc_size + threadsPerBlock - 1) / threadsPerBlock;

	antenna_init_kernel <<< blocksPerGrid, threadsPerBlock >>> (
		d_malloc_size, current.lambda, current.spacing, current.theta_c, current.panel_count, current.antenna_dims,
		d_phee_minus_alpha_list, d_pathloss_list, d_gain_RX_grid, d_polar_data);

	hipError_t cudaStatus = hipGetLastError();

	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "antenna_init_kernel() launch failed: %s\n", hipGetErrorString(cudaStatus));
		exit(-1);
	}

	/* hipDeviceSynchronize waits for the kernel to finish, and returns
	 any errors encountered during the launch. */
	cudaStatus = hipDeviceSynchronize();

	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		exit(-1);
	}
}

/* update the antenna array from updated power and scan angle */
__host__ void AAntenna::graphics_update()
{
	if (graphic.modified)
	{
		update(d_malloc_size_gfx, d_phee_minus_alpha_list_gfx, d_gain_RX_grid_gfx, d_pathloss_list_gfx, d_hmatrix_gfx, host_hmatrix_gfx);
		graphic.modified = false;
	}
}

__host__ void AAntenna::numerical_update()
{
	if (simulation.modified)
	{
		update(d_malloc_size_sim, d_phee_minus_alpha_list_sim, d_gain_RX_grid_sim, d_pathloss_list_sim, d_hmatrix_sim, host_hmatrix_sim);
		simulation.modified = false;
	}
}


/* for GUI simulation in the whole grid */
__host__ void AAntenna::graphics_init(PolarArray& polar_data)
{
	if (d_malloc_size_sim == 0 || d_malloc_size_sim != polar_data.array_size)
	{
		CUDA_GFREE();
		CUDA_GINIT(polar_data.array_size);
	}

	graphic.host_hmatrix = host_hmatrix_gfx;
	host_polar_gfx = polar_data.data_ptr;

	hipStream_t stream;
	hipStreamCreate(&stream);

	hipMemcpyAsync(d_polar_data_gfx, host_polar_gfx, polar_data.array_size * sizeof(Polar_Coordinates), hipMemcpyHostToDevice, stream);

	hipStreamSynchronize(stream);
	hipStreamDestroy(stream);
	//hipMemcpy(d_polar_data_gfx, host_polar_gfx, polar_data.array_size * sizeof(Polar_Coordinates), hipMemcpyHostToDevice);

	init(d_malloc_size_gfx, d_phee_minus_alpha_list_gfx, d_pathloss_list_gfx, d_gain_RX_grid_gfx, d_polar_data_gfx);
	graphic.modified = true;
}


/* for bare-minimum numerical calculations needed at the mobile_stations only */
__host__ void AAntenna::numerical_init(PolarArray& polar_data)
{
	if (d_malloc_size_sim == 0 || d_malloc_size_sim != polar_data.array_size)
	{
		CUDA_FREE();
		CUDA_INIT(polar_data.array_size);
	}

	simulation.host_hmatrix = host_hmatrix_sim;
	host_polar_sim = polar_data.data_ptr;

	hipStream_t stream;
	hipStreamCreate(&stream);

	hipMemcpyAsync(d_polar_data_sim, host_polar_sim, polar_data.array_size * sizeof(Polar_Coordinates), hipMemcpyHostToDevice, stream);

	hipStreamSynchronize(stream);
	hipStreamDestroy(stream);
	//hipMemcpy(d_polar_data_sim, polar_data.data_ptr, polar_data.array_size * sizeof(Polar_Coordinates), hipMemcpyHostToDevice);

	init(d_malloc_size_sim, d_phee_minus_alpha_list_sim, d_pathloss_list_sim, d_gain_RX_grid_sim, d_polar_data_sim);
	simulation.modified = true;
}


__host__ AAntenna::~AAntenna()
{
	hipError_t cudaStatus = hipDeviceSynchronize();

	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize returned error code %d in Destructor of Antenna object!\n", cudaStatus);
		exit(-1);
	}

	CUDA_GFREE();
	CUDA_FREE();
}

/* hatrix with respect to pixel index (flattened from 2D) */
const double& AAntenna::gcoeff(const unsigned& pixel_idx) const
{
	return graphic.host_hmatrix[pixel_idx];
}

const double& AAntenna::coeff(const unsigned& rx_sta) const
{
	return simulation.host_hmatrix[rx_sta];
}

#endif
